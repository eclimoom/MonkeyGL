#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>


extern "C"
void cu_copyVolumeData( short* h_volumeData, hipExtent volumeSize, hipArray*& d_volumeArray, hipTextureObject_t& volumeTexture)
{
	if (d_volumeArray != 0)
	{
		checkCudaErrors(hipFreeArray(d_volumeArray));
		d_volumeArray = 0;
		volumeTexture = 0;
	}

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<short>();
	checkCudaErrors( hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize) );

	hipMemcpy3DParms copyParams = {0};
	copyParams.dstArray = d_volumeArray;
	copyParams.extent   = volumeSize;
	copyParams.kind     = hipMemcpyHostToDevice;
	copyParams.srcPtr   = make_hipPitchedPtr(
		(void*)h_volumeData,
		volumeSize.width*sizeof(short),
		volumeSize.width,
		volumeSize.height
	);

	checkCudaErrors( hipMemcpy3D(&copyParams) );  
	
	hipResourceDesc texRes;
	memset(&texRes, 0, sizeof(hipResourceDesc));

	texRes.resType = hipResourceTypeArray;
	texRes.res.array.array = d_volumeArray;

	hipTextureDesc texDescr;
	memset(&texDescr, 0, sizeof(hipTextureDesc));

	texDescr.normalizedCoords = true;  // access with normalized texture coordinates
	texDescr.filterMode = hipFilterModeLinear;  // linear interpolation

	texDescr.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
	texDescr.addressMode[1] = hipAddressModeClamp;
	texDescr.addressMode[2] = hipAddressModeClamp;

	texDescr.readMode = hipReadModeNormalizedFloat;
		
	checkCudaErrors( hipCreateTextureObject(&volumeTexture, &texRes, &texDescr, NULL) );
}

extern "C"
void cu_copyMaskData(unsigned char* h_maskData, hipExtent volumeSize, hipArray*& d_maskArray, hipTextureObject_t& maskTexture)
{
	if (d_maskArray != 0)
	{
		checkCudaErrors(hipFreeArray(d_maskArray));
		d_maskArray = 0;
	}

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned char>();
	checkCudaErrors( hipMalloc3DArray(&d_maskArray, &channelDesc, volumeSize) );

	hipMemcpy3DParms copyParams = {0};
	copyParams.dstArray = d_maskArray;
	copyParams.extent   = volumeSize;
	copyParams.kind     = hipMemcpyHostToDevice;
	copyParams.srcPtr   = make_hipPitchedPtr(
		(void*)h_maskData,
		volumeSize.width*sizeof(unsigned char),
		volumeSize.width,
		volumeSize.height
	);

	checkCudaErrors( hipMemcpy3D(&copyParams) );  
	
	hipResourceDesc texRes;
	memset(&texRes, 0, sizeof(hipResourceDesc));

	texRes.resType = hipResourceTypeArray;
	texRes.res.array.array = d_maskArray;

	hipTextureDesc texDescr;
	memset(&texDescr, 0, sizeof(hipTextureDesc));

	texDescr.normalizedCoords = false;
	texDescr.filterMode = hipFilterModePoint; 

	texDescr.addressMode[0] = hipAddressModeClamp;
	texDescr.addressMode[1] = hipAddressModeClamp;
	texDescr.addressMode[2] = hipAddressModeClamp;

	texDescr.readMode = hipReadModeElementType;
		
	checkCudaErrors( hipCreateTextureObject(&maskTexture, &texRes, &texDescr, NULL) );
}

extern "C"
void cu_setTransferFunc( float* pTransferFunc, int nLenTransferFunc, hipArray*& d_transferFuncArray, hipTextureObject_t& transferFuncTexture)
{
	hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));
    texRes.resType = hipResourceTypeArray;

    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));
    texDescr.normalizedCoords = true;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeClamp;
    texDescr.readMode = hipReadModeElementType;

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();

    if (d_transferFuncArray != 0)
	{
		checkCudaErrors(hipFreeArray(d_transferFuncArray));
		d_transferFuncArray = 0;
	}
    checkCudaErrors(hipMallocArray( &d_transferFuncArray, &channelDesc, nLenTransferFunc, 1));
    checkCudaErrors(
        hipMemcpy2DToArray(
            d_transferFuncArray, 
            0, 
            0, 
            pTransferFunc,
            0, 
            nLenTransferFunc*sizeof(float4), 
            1,
            hipMemcpyHostToDevice
        )
    );

    texRes.res.array.array = d_transferFuncArray;

    checkCudaErrors(
        hipCreateTextureObject(&transferFuncTexture, &texRes, &texDescr, NULL)
    );
}